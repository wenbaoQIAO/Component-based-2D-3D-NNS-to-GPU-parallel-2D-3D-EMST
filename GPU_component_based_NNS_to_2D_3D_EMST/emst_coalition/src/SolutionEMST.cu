#include "hip/hip_runtime.h"
﻿#include "config/ConfigParamsCF.h"
#include "random_generator_cf.h"
#include "CalculateurEMST.h"
//#include "Multiout.h"

#include "SolutionEMST.h"

#define round(x) ((fabs(ceil(x) - (x)) < fabs(floor(x) - (x))) ? ceil(x) : floor(x))

#define EMST_PRINT_CM 1
#define EMST_SQUARED_CM 0

template<std::size_t DimP, std::size_t DimCM>
int SolutionEMST<DimP, DimCM>::cptInstance = 0;

template<std::size_t DimP, std::size_t DimCM>
void SolutionEMST<DimP, DimCM>::initialize(char* data, char* sol, char* stats)
{
    fileData = data;
    fileSolution = sol;
    fileStats = stats;

    initialize();
}

template<std::size_t DimP, std::size_t DimCM>
void SolutionEMST<DimP, DimCM>::initialize()
{
    printf(" CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        // Console log
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

        char msg[256];
        SPRINTF(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        printf("%s", msg);

        printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
               deviceProp.multiProcessorCount,
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
        printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);


#if CUDART_VERSION >= 5000
        // This is supported in CUDA 5.0 (runtime API device properties)
        printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
        printf("  Memory Bus Width:                              %d-bit\n",   deviceProp.memoryBusWidth);

        if (deviceProp.l2CacheSize)
        {
            printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
        }

#else
        // This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
        int memoryClock;
        getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
        printf("  Memory Clock rate:                             %.0f Mhz\n", memoryClock * 1e-3f);
        int memBusWidth;
        getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
        printf("  Memory Bus Width:                              %d-bit\n", memBusWidth);
        int L2CacheSize;
        getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

        if (L2CacheSize)
        {
            printf("  L2 Cache Size:                                 %d bytes\n", L2CacheSize);
        }

#endif

        printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
               deviceProp.maxTexture1D   , deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
               deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
        printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
               deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
        printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
               deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);


        printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
        printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n", deviceProp.warpSize);
        printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
        printf("  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
        printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
        printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
        printf("  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
        printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
        printf("  Alignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
        printf("  Device has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
        printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
        printf("  Device supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
        printf("  Supports Cooperative Kernel Launch:            %s\n", deviceProp.cooperativeLaunch ? "Yes" : "No");
        printf("  Supports MultiDevice Co-op Kernel Launch:      %s\n", deviceProp.cooperativeMultiDeviceLaunch ? "Yes" : "No");
        printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);

        const char *sComputeMode[] =
        {
            "Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
            "Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
            "Prohibited (no host thread can use ::hipSetDevice() with this device)",
            "Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
            "Unknown",
            NULL
        };
        printf("  Compute Mode:\n");
        printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
    }

    // If there are 2 or more GPUs, query to determine whether RDMA is supported
    if (deviceCount >= 2)
    {
        hipDeviceProp_t prop[64];
        int gpuid[64]; // we want to find the first two GPUs that can support P2P
        int gpu_p2p_count = 0;

        for (int i=0; i < deviceCount; i++)
        {
            checkCudaErrors(hipGetDeviceProperties(&prop[i], i));

            // Only boards based on Fermi or later can support P2P
            if ((prop[i].major >= 2)
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
                // on Windows (64-bit), the Tesla Compute Cluster driver for windows must be enabled to support this
                && prop[i].tccDriver
#endif
               )
            {
                // This is an array of P2P capable GPUs
                gpuid[gpu_p2p_count++] = i;
            }
        }

        // Show all the combinations of support P2P GPUs
        int can_access_peer;

        if (gpu_p2p_count >= 2)
        {
            for (int i = 0; i < gpu_p2p_count; i++)
            {
                for (int j = 0; j < gpu_p2p_count; j++)
                {
                    if (gpuid[i] == gpuid[j])
                    {
                        continue;
                    }
                    checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer, gpuid[i], gpuid[j]));
                        printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[i]].name, gpuid[i],
                           prop[gpuid[j]].name, gpuid[j] ,
                           can_access_peer ? "Yes" : "No");
                }
            }
        }
    }

    // csv masterlog info
    // *****************************
    // exe and CUDA driver name
    printf("\n");
    std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
    char cTemp[16];

    // driver version
    sProfileString += ", CUDA Driver Version = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    sprintf_s(cTemp, 10, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#else
    sprintf(cTemp, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Runtime version
    sProfileString += ", CUDA Runtime Version = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    sprintf_s(cTemp, 10, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#else
    sprintf(cTemp, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Device count
    sProfileString += ", NumDevs = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    sprintf_s(cTemp, 10, "%d", deviceCount);
#else
    sprintf(cTemp, "%d", deviceCount);
#endif
    sProfileString += cTemp;
    sProfileString += "\n";
    printf("%s", sProfileString.c_str());

    printf("Result = PASS\n");

#if TEST_CODE
        int devID = 0;
        hipError_t error;
        hipDeviceProp_t deviceProp;
        error = hipGetDevice(&devID);
        if (error != hipSuccess)
        {
            printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
        }
        error = hipGetDeviceProperties(&deviceProp, devID);
        if (deviceProp.computeMode == hipComputeModeProhibited)
        {
            fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
            exit(EXIT_SUCCESS);
        }
        if (error != hipSuccess)
        {
            printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        }
        else
        {
            printf("GPU Device %d: \"%s\" with compute capability %d.%d\n",
                   devID, deviceProp.name, deviceProp.major, deviceProp.minor);
            printf("GPU Device %d: \"%s\" with multi-processors %d\n",
                   devID, deviceProp.name, deviceProp.multiProcessorCount);
        }
#endif
}

template<std::size_t DimP, std::size_t DimCM>
void SolutionEMST<DimP, DimCM>::initialize(NetLink& md_links, PointEuclidean& pMin, PointEuclidean& pMax)
{
    cout << "INITIALISATION START" << std::endl;

    size_t nNodes = 0;
    GLint w = md_links.adaptiveMap.getWidth();
    GLint h = md_links.adaptiveMap.getHeight();
    GLint d = md_links.adaptiveMap.getDepth();
    nNodes = w * h * d;
    cout << "num cities = " << nNodes  << endl;

    // Save original map
    adaptiveMapOriginal.gpuResize(w, h, d);
    md_links.adaptiveMap.gpuCopyHostToDevice(adaptiveMapOriginal);

    // Change coordinates
#if EMST_SQUARED_CM
    GLdouble min_x = pMin[0];
    GLdouble min_y = pMin[1];
    GLdouble max_x = pMax[0];
    GLdouble max_y = pMax[1];

    int _w = (GLint) ceil(max_x - min_x)+2;
    int _h = (GLint) ceil(max_y - min_y)+2;
    cout << "city_area_w , city_area_h " << _w << ", " << _h << endl;

    // wb.Q, chansfer the coordinate system
    float unitX = sqrt(nNodes) / _w;
    float unitY = sqrt(nNodes) / _h;
    cout << "unitX = " << unitX << endl;
    cout << "unitY = " << unitY << endl;
#else
    GLdouble wg = pMax[0] - pMin[0];
    GLdouble hg = pMax[1] - pMin[1];
    GLdouble dg = 0;
    if (DimP >= 3)
        dg = pMax[2] - pMin[2];

//    if (wg == 0 && hg == 0 && dg = 0) {
//        wg = hg = dg = 1;
//    }

    cout << "max original x, y, z " << pMax[0] << ", " << pMax[1];
    if (DimP >= 3)
        cout << ", " << pMax[2];
    cout << endl;
    cout << "min original x, y, z " << pMin[0] << ", " << pMin[1];
    if (DimP >= 3)
        cout << ", " << pMin[2];
    cout << endl;
    cout << "city_area : " << wg << " * " << hg << " * " << dg << endl;

    if (wg == 0)
        wg = 1;
    if (hg == 0)
        hg = 1;
    GLdouble _wnd = sqrt(nNodes*wg/hg)*2;
    if (DimP >= 3) {
        if (dg != 0)
            _wnd = pow((nNodes*wg*wg)/(hg*dg), 0.333);
        else
            dg = 1;
    }
    //GLdouble _hnd = sqrt(nNodes*hg/wg);

    GLdouble _wndd = ceil(_wnd) + 1;
    //GLdouble _hndd = ceil(_hnd) + 1;

    GLdouble unitX = _wndd / wg;
    //GLdouble unitY = unitX;//_hndd / hg;
    cout << "unitX = " << unitX << endl;
    //cout << "unitY = " << unitY << endl;

//    GLint _wn = (GLint) _wndd + 1;
//    GLint _hn = (GLint) _hndd + 1;
#endif
    // Change coord system
    mr_links_cpu.resize(nNodes, 1);
    GLdouble max_xNew = -INFINITY;
    GLdouble max_yNew = -INFINITY;
    GLdouble min_xNew = +INFINITY;
    GLdouble min_yNew = +INFINITY;
    GLdouble max_zNew = -INFINITY;
    GLdouble min_zNew = +INFINITY;
    if (DimP < 3) {
        max_zNew = 0;
        min_zNew = 0;
    }
    IndexG idx(0);
    mr_links_cpu.adaptiveMap.iterInit(idx);
    while (mr_links_cpu.adaptiveMap.iterNext(idx)) {
        mr_links_cpu.adaptiveMap(idx) = (md_links.adaptiveMap(idx) - pMin) * unitX;// float * double
        if (mr_links_cpu.adaptiveMap(idx)[0] >= max_xNew) // float vs double
            max_xNew = mr_links_cpu.adaptiveMap(idx)[0];
        if (mr_links_cpu.adaptiveMap(idx)[1] >= max_yNew)
            max_yNew = mr_links_cpu.adaptiveMap(idx)[1];
        if (mr_links_cpu.adaptiveMap(idx)[0] < min_xNew)
            min_xNew = mr_links_cpu.adaptiveMap(idx)[0];
        if (mr_links_cpu.adaptiveMap(idx)[1] < min_yNew)
            min_yNew = mr_links_cpu.adaptiveMap(idx)[1];
        if (DimP >= 3) {
            if (mr_links_cpu.adaptiveMap(idx)[2] >= max_zNew)
                max_zNew = mr_links_cpu.adaptiveMap(idx)[2];
            if (mr_links_cpu.adaptiveMap(idx)[2] < min_zNew)
                min_zNew = mr_links_cpu.adaptiveMap(idx)[2];
        }
    }

    cout << "max x,y,z New " << max_xNew << ", " << max_yNew << ", " << max_zNew << endl;
    cout << "min x,y,z New " << min_xNew << ", "  << min_yNew << ", " << min_zNew << endl;

    mr_links_gpu.gpuResize(w, h);
    distanceMap_cpu.resize(w, h, d);
    distanceMap.gpuResize(w, h, d);

    mr_links_cpu.evtMap.resize(w, h, d);
    mr_links_gpu.evtMap.gpuResize(w, h, d);
    mr_links_cpu.nVisitedMap.resize(w, h, d);
    mr_links_gpu.nVisitedMap.gpuResize(w, h, d);
    mr_links_cpu.nodeParentMap.resize(w, h, d);
    mr_links_gpu.nodeParentMap.gpuResize(w, h, d);
    mr_links_cpu.nodeWinMap.resize(w, h, d);
    mr_links_gpu.nodeWinMap.gpuResize(w, h, d);
    mr_links_cpu.nodeDestMap.resize(w, h, d);
    mr_links_gpu.nodeDestMap.gpuResize(w, h, d);

    minDistMap_cpu.resize(w, h, d);
    minDistMap.gpuResize(w, h, d);
    stateMap_cpu.resize(w, h, d);
    stateMap.gpuResize(w, h, d);

    spiralSearchMap_cpu.resize(w, h, d);
    spiralSearchMap.gpuResize(w, h, d);

    mr_links_cpu.gpuCopyHostToDevice(mr_links_gpu);

    // Cellular matrix creation
    ExtentsCM ext(1);
#if EMST_SQUARED_CM
    int _wn = int(max_xNew - min_xNew) + 3;
    int _hn = int(max_yNew - min_yNew) + 3;
    ext[0] = (GLint) ceil(max_xNew - min_xNew);
    ext[1] = (GLint) ceil(max_yNew - min_yNew);
    if (DimCM >= 3)
        ext[2] = (GLint) ceil(max_zNew - min_zNew)+1;
    cout << "vgd area _w,_h " << _wn << ", " << _hn << endl;
#else
    ext[0] = (GLint) ceil(max_xNew - min_xNew)+1;
    ext[1] = (GLint) ceil(max_yNew - min_yNew)+1;
    if (DimCM >= 3)
        ext[2] = (GLint) ceil(max_zNew - min_zNew)+1;
//    ext[0] = _wn;
//    ext[1] = _hn;
#endif
    IndexCM pc;
    pc = ext / 2;
    int _R = g_ConfigParameters->levelRadius;
    vgd = ViewG(pc, ext, _R);

    cout << "vgd dual : "
         << vgd.getExtentsDual() << endl
         << "vgd base : "
         << vgd.getExtentsBase() << endl
         << "vgd low level : "
         << vgd.getExtents() << endl;
    cout << "nNodes " << nNodes << ", sqrt(nNodes) " << sqrt(nNodes) << endl;

    // Cellular matrix initialisations
    cm_gpu.setViewG(vgd);
    cm_gpu.gpuResize(vgd.getExtentsDual());
    cm_gpu.K_initialize(vgd);

    cout << "CM GPU RESIZE DONE " << cm_gpu.length_in_bytes << " " << cm_gpu.length << endl;

    cm_cpu.setViewG(vgd);
    cm_cpu.resize(vgd.getExtentsDual());
    //cm_cpu.K_initialize_cpu(vgd);

    cout << "CM RESIZE DONE " << cm_cpu.length_in_bytes << " " << cm_cpu.length << endl;

    iteration = 0;

    // Initialze mr_links attributes
    boruvkaOp.gpuResetValue(mr_links_gpu);
    boruvkaOp.cpuResetValue(mr_links_cpu);

    // Initialize disjoint set structure
    boruvkaOp.K_initDisjointSet(mr_links_gpu.disjointSetMap);

    boruvkaOp.K_clearLinks(mr_links_gpu.networkLinks);

    distanceMap.gpuResetValue(HUGE_VAL);
    PointCoord pInitial(-1);
    mr_links_gpu.correspondenceMap.gpuResetValue(pInitial);

    // Kernel Time init
    time_next_closest = 0;
    time_find_pair = 0;
    time_connect_union = 0;
    time_flatten = 0;
    time_terminate = 0;

    cm_gpu.K_clearCells();

    //boruvkaOp.K_refreshCell(cm_gpu, mr_links_gpu.adaptiveMap);

    // Spiral search Grid initialisation
    boruvkaOp.K_initializeSpiralSearch(cm_gpu,
                                       mr_links_gpu.adaptiveMap,
                                       spiralSearchMap);

    cout << "CM GPU K_initializeSpiralSearch DONE" << endl;
#if EMST_PRINT_CM
    cm_cpu.gpuCopyDeviceToHost(cm_gpu);
    cout << "CM gpuCopyDeviceToHost DONE" << endl;
    int numNode = 0;
    int maxSize = 0;
    IndexCM idxcm(0);
    cm_cpu.iterInit(idxcm);
    while (cm_cpu.iterNext(idxcm)) {
        if (cm_cpu(idxcm).size > 0
                && cm_cpu(idxcm).size > MAX_CELL_SIZE // wb.Q tests which cell exceeds the maxmum bound
                ) {
            cout << "cmd_cpu " <<  idxcm << endl;
            cout << " " << cm_cpu(idxcm).size << endl;
        }
        numNode += cm_cpu(idxcm).size;
        maxSize = (cm_cpu(idxcm).size > maxSize) ? cm_cpu(idxcm).size : maxSize;
    }
    cout << "=== check num of nodes inserted into cmd: " << numNode << " max cell size: "<< maxSize << endl;
#endif
    cout << "INITIALISATION DONE" << std::endl;


//    (ofstream&)cout << * (CalculateurEMST::getAdaptiveMap3()) << endl;

}

template<std::size_t DimP, std::size_t DimCM>
void SolutionEMST<DimP, DimCM>::clone(SolutionEMST* sol)
{
    (*sol).fileData = (*this).fileData;
    (*sol).fileSolution = (*this).fileSolution;
    (*sol).fileStats = (*this).fileStats;

    (*sol).t0 = (*this).t0;
    (*sol).tf = (*this).tf;
    (*sol).x0 = (*this).x0;
    (*sol).xf = (*this).xf;

#ifdef CUDA_CODE
    (*sol).start = (*this).start;
    (*sol).stop = (*this).stop;
#endif
    (*sol).global_objectif = (*this).global_objectif;

    (*this).md_links_cpu.clone((*sol).md_links_cpu);

    // size numbers
    (*sol).pMin = (*this).pMin;
    (*sol).pMax = (*this).pMax;

    (*sol).initialize((*sol).md_links_cpu, (*sol).pMin, (*sol).pMax);

}

template<std::size_t DimP, std::size_t DimCM>
void SolutionEMST<DimP, DimCM>::setIdentical(SolutionEMST* sol)
{
    (*sol).global_objectif = (*this).global_objectif;

    (*this).mr_links_cpu.setIdentical((*sol).mr_links_cpu);
    (*this).mr_links_gpu.gpuSetIdentical((*sol).mr_links_gpu);

    (*sol).traceParallelMST = (*this).traceParallelMST;

}

template<std::size_t DimP, std::size_t DimCM>
void SolutionEMST<DimP, DimCM>::initEvaluate()
{
    traceParallelMST.length = 0;//numeric_limits<double>::max();
    traceParallelMST.size = 0;//numeric_limits<double>::max();
    this->global_objectif = 0;//numeric_limits<double>::max();
}

template<std::size_t DimP, std::size_t DimCM>
double SolutionEMST<DimP, DimCM>::evaluate()
{
    //-------------------------------------------------------------------------
    // Mise à jour positions vehicules, chemins, volumes
    //-------------------------------------------------------------------------
    initEvaluate();

    boruvkaOp.K_evaluate_ST(mr_links_gpu.networkLinks, adaptiveMapOriginal, mr_links_gpu.objectivesMap);
    mr_links_cpu.objectivesMap.gpuCopyDeviceToHost(mr_links_gpu.objectivesMap);
    AMObjectives obj(0);
    BOp op;
    op.K_sumReduction(mr_links_cpu.objectivesMap, obj);
    traceParallelMST.size = obj[obj_distr]/2;
    traceParallelMST.length = obj[obj_length]/2;

    cout << "EMST SIZE .................. " << traceParallelMST.size << endl;
    cout << "EMST LENGTH .................. " << traceParallelMST.length << endl;
    // Calcul objectif global
    computeObjectif();

    return global_objectif;
}//evaluate

/*!
 * \return valeur de la fonction objectif agregative
 */
template<std::size_t DimP, std::size_t DimCM>
double SolutionEMST<DimP, DimCM>::computeObjectif(void)
{
    global_objectif = traceParallelMST.size;//traceParallelMST.length;

    return global_objectif;
}

/*!
 * \param best SolutionEMST comparee
 * \return vrai si objectif de l'appelant (ie la SolutionEMST courante) est inferieur ou egal a celui de la SolutionEMST comparee
 */
template<std::size_t DimP, std::size_t DimCM>
bool SolutionEMST<DimP, DimCM>::isBest(SolutionEMST* best)
{
    bool res = false;

    if (computeObjectif() <= best->computeObjectif())
        res = true;

    return res;
}

/*!
 * \return vrai si SolutionEMST admissible
 */
template<std::size_t DimP, std::size_t DimCM>
bool SolutionEMST<DimP, DimCM>::isSolution()
{
    bool res = false;
    if (this->global_objectif <= 0)    {
        res = true;
    }
    return res;
}//isSolutionEMST

